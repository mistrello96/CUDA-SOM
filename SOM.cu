#include "hip/hip_runtime.h"
//#include <stdio.h>
//#include <iostream>
//#include <hip/hip_runtime.h>
//#include <string>
//#include <cmath>
#include <ctime>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <unistd.h>
#include <fstream>
#include <sstream>
#include <float.h>

#define CUDA_CHECK_RETURN(value) {											\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(-13);															\
} }


__global__ void compute_distance(double* k_matrix, double* k_ActualSample, double* k_distance, int nNeuron, int nElements){
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index < nNeuron)
	{
		int matrixindex = index * nElements;
		double tmp = 0;
		for(int i = 0; i < nElements; i++)
		{
			tmp = tmp + powf(k_matrix[matrixindex+i] - k_ActualSample[i], 2.0);
		}

		k_distance[index] = sqrtf(tmp);
		//printf("%f\n", k_distance[index]);
	}
}


double checkFreeGpuMem(){
	double free_m;
	size_t free_t,total_t;
	hipMemGetInfo(&free_t,&total_t);
	free_m =(uint)free_t;
	return free_m;
}


// returns the number of features per line
int readSamplesfromFile(std::vector<double>& samples, std::string filePath){
	int counter = 0;
	std::string line;
	std::ifstream file (filePath.c_str());
	if (file.is_open()) {
		while (std::getline (file, line)) {
			std::istringstream iss(line);
    		std::string element;
    		int tmp = 0;
    		while(std::getline(iss, element, '\t')){
    			tmp ++;
				samples.push_back(strtof((element).c_str(),0));
			}
			counter = tmp;
		}
		file.close();
		return counter;
	}
	else{
		std::cout << "Unable to open file";
		exit(-1);
	}
}


int main(int argc, char **argv)
{
	// INIZIALIZING VARIABLES WITH DEFAULT VALUES
	// path of the input file
	std::string filePath = "./";
	// debuf flag
    bool debug = false;
    // number of rows in the martix
    int nRows = 0;
    // number of column in the martix
    int nColumns = 0;
    // initial learning rate
    double ilr = 0;
    // final learning rate
    double flr = 0;
    // lambda of the gaussian
    double lambda = 0;
    // max number of iteration
    int maxnIter = 0;
    // accuracy threshold
    double requiredAccuracy = 0;
    // learning iteration counter
    int nIter = 0;

    // COMMAND LINE PARSING
    int c;
    while ((c = getopt (argc, argv, "i:x:y:hvs:f:m:a:l:")) != -1)
    switch (c)
	{
        case 'i':
            filePath = optarg;
            break;
        case 'n':
            if (int (sqrt(atoi(optarg))) * int (sqrt(atoi(optarg))) != atoi(optarg)){
                std::cout << "The -x option only support square matrix. To create a generic matrix, use -x and -y parameters" << std::endl;
                return(-1);
            }
            nRows = sqrt(atoi(optarg));
            nColumns = sqrt(atoi(optarg));
            break;
        case 'x':
            nRows = atoi(optarg);
            break;
        case 'y':
            nColumns = atoi(optarg);
            break;
        case 's':
            ilr = strtof(optarg,0);
            break;
        case 'f':
            flr = strtof(optarg,0);
            break;
        case 'a':
            requiredAccuracy = strtof(optarg,0);
            break;
        case 'l':
            lambda = strtof(optarg,0);
            break;
        case 'm':
            maxnIter = atoi(optarg);
            break;
        case 'v':
            debug = true;
            break;
        case 'h':
            std::cout << "-i allows to provide the PATH of an input file. If not specified, ./ is assumed" << std::endl;
            std::cout << "-x allows to provide the number of rows in the neuron's matrix. REQUESTED" << std::endl;
            std::cout << "-y allows to provide the numbers of columns in the neuron's matrix. REQUESTED" << std::endl;
            std::cout << "-s initial learning rate" << std::endl;
            std::cout << "-f final learning rate" << std::endl;
            std::cout << "-l lambda of the gaussian" << std::endl;
            std::cout << "-a accuracy threshold" << std::endl;
            std::cout << "-m maximum number of iteration before stopping the learning process" << std::endl;
            std::cout << "-v enables debug prints" << std::endl;
            std::cout << "-h shows help menu of the tool" << std::endl;
            return 0;
    }

    // checking the required params
    if (nRows == 0 | nColumns == 0 | ilr == 0 | maxnIter == 0){
        std::cout << "Required params are missing, program will abort" << std::endl;
        exit(-1);        
    }

    // READ THE INPUT FILE
    // vector of samples to be analized from the SOM
    std::vector <double> Samples;
    // retrive the number of features readed from the file
    int nElements = readSamplesfromFile(Samples, filePath);

    // EXTRACTING THE MIN/MAX FROM SAMPLES
    // creating the thrust vector
    thrust::device_vector<double> t_Samples(Samples);
    // extract the minimum
    thrust::device_vector<double>::iterator it = thrust::min_element(t_Samples.begin(), t_Samples.end());
    double min_neuronValue = *it;
    // extract maximum
    thrust::device_vector<double>::iterator it2 = thrust::max_element(t_Samples.begin(), t_Samples.end());
    double max_neuronValue = *it2;

    // COMPUTE USEFULL VALUES
    // total number of neurons in the SOM
    int nNeurons = nRows * nColumns;
    // total length of the serialized matrix
    int totalLength = nRows * nColumns * nElements;
    // number of block used in the computation
    int nblocks = (nNeurons / 1024) + 1;
    // inizializing the learnig rate
    double lr = ilr;
    // retrive the number of samples
    int nSamples = Samples.size() / nElements;

    // CHECKING COMPUTABILITY ON CUDA
    if (nblocks >= 65535){
    	std::cout << "Too many bocks generated, cannot run a kernel with so many blocks. Try to reduce the number of neurons" << std::endl;
    	exit(-1);
    }

    // CHECK AVAILABLE MEMORY
    if (sizeof(double) * nNeurons * nElements >= checkFreeGpuMem()){
	    	std::cout << "Not enougth memory on the GPU, try to reduce neurons' number" << std::endl;
	    	exit(-1);
	}

    // debug print
    if(debug){
        std::cout << "Running the program with " << nRows  << " rows, " << nColumns << " columns, " << nNeurons << " neurons, " << nElements << " features." << std::endl;
    }

    // ALLOCATION OF THE STRUCTURES
    // host SOM
    double *h_Matrix = (double *)malloc(sizeof(double) * totalLength);
    // host sample array
    double *h_ActualSample = (double *)malloc(sizeof(double) * nElements);
    // host distance array, used to find BMU
    double *h_Distance = (double *) malloc(sizeof(double) * nNeurons);
    // host BMU distance array
    double *h_DistanceHistory = (double *)malloc(sizeof(double) * nSamples);
    // device SOM
    double *d_Matrix;
    // device sample array
    double *d_Sample;
    // device distance array, 
    double *d_Distance;
    // device malloc
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_Matrix, sizeof(double) * totalLength));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_Sample, sizeof(double) * nElements));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_Distance, sizeof(double) * nNeurons));

    // SOM INIZIALIZATION
    // generating random seed
    srand(time(NULL));
    // random values SOM initialization
    for(int i = 0; i < totalLength; i++){
    	double tmp = rand() / (float) RAND_MAX;
    	tmp = min_neuronValue + tmp * (max_neuronValue - min_neuronValue);
    	h_Matrix[i] = tmp; 
    }

    double accuracy = DBL_MAX;
    while((accuracy > requiredAccuracy) && (lr > flr) && (nIter < maxnIter)){
    	// TODO Randomize sample vector
	    // ITERATE ON EACH SAMPLE TO FIND BMU
	    for(int s=0; s < nSamples ; s++){

		    // distance array inizialization
		    for(int i = 0; i < nNeurons; i++){
		    	h_Distance[i] = 0;
		    }

		    // copy the s sample in the actual sample vector
		    for(int i = s*nElements, j = 0; i < s*nElements+nElements; i++, j++){
		    	h_ActualSample[j] = Samples[i];
		    }; 

			// copy from host to device matrix, actual sample and distance
			CUDA_CHECK_RETURN(hipMemcpy(d_Matrix, h_Matrix, sizeof(double) * totalLength, hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy(d_Sample, h_ActualSample, sizeof(double) * nElements, hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy(d_Distance, h_Distance, sizeof(double) * nNeurons, hipMemcpyHostToDevice));	
			
		    // parallel search launch
		    compute_distance<<<nblocks,1024>>>(d_Matrix, d_Sample, d_Distance, nNeurons, nElements);

			//wait for all block to complete the computation
		    hipDeviceSynchronize();

		    // CHECK AVAILABLE MEMORY
	    	if (sizeof(double) * nNeurons * nElements >= checkFreeGpuMem()){
		    	std::cout << "Out of memory, try to reduce the neurons number" << std::endl;
		    	exit(-1);
			}
				
			// create thrust vector to find BMU  in parallel
			thrust::device_vector<double> d_vec_Distance(d_Distance, d_Distance + nNeurons);
			// extract the first matching BMU
			thrust::device_vector<double>::iterator iter = thrust::min_element(d_vec_Distance.begin(), d_vec_Distance.end());
			// extract index and value of BMU
			unsigned int BMU_index = iter - d_vec_Distance.begin();
			double BMU_distance = *iter;
			// adding the found value in the distance history array
			h_DistanceHistory[s] = BMU_distance;

			// debug print
		    if(debug)
			   std::cout << "The minimum distance is " << BMU_distance << " at position " << BMU_index << std::endl;

	        //TODO: update BMU and neighbors
	        
	        for (int i = BMU_index * nElements, j = 0; j < nElements; i++, j++){
	        	h_Matrix[i] = h_Matrix[i] + lr*(h_ActualSample[j] - h_Matrix[i]);
	        }
	         
		}

		if (debug){
			std::cout << "Learn rate of this iteration is " << lr << std::endl;
		}
	
		// updating the counter iteration
		nIter ++;
		// updating the learning rate
		lr = ilr - 0.01*nIter;
		// updating accuracy
		thrust::device_vector<double> d_DistanceHistory(h_DistanceHistory, h_DistanceHistory + nSamples);
		double meansum = thrust::reduce(d_DistanceHistory.begin(), d_DistanceHistory.end());
		accuracy = meansum / ((double)nSamples);

		if (debug){
			std::cout << "Mean distance of this iteration is " << accuracy << std::endl;
		}

	}

	//freeing all allocated memory
    hipFree(d_Matrix);
    hipFree(d_Sample);
    hipFree(d_Distance);
    free(h_Matrix);
    free(h_Distance);
    free(h_ActualSample);

}

