#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <unistd.h>
#include <cmath>

__global__ void
compute_distance(float* k_matrix, int nNeuron, float* k_sample, float* k_distance, int sampleLength)
{
	int index = (threadIdx.x + blockIdx.x * blockDim.x);
	if (index < nNeuron)
	{
		int matrixindex = index * 14;
		float tmp = 0;
		for(int i = 0; i < sampleLength; i++)
		{
			tmp = tmp + abs(k_matrix[matrixindex+i] - k_sample[i]);
		}

		k_distance[index] = k_distance[index] + tmp;
	}
}

int
main(int argc, char **argv)
{
	// number of features in each neuron
    int nElements = 14;
    // number of rows in the martix
    int nRows = 1000;
    // number of column in the martix
    int nColumns = 1000;

    //command line parsing
    int c;
    while ((c = getopt (argc, argv, "i:n:x:y:")) != -1)
    switch (c) {
        case 'i':
            //filepath = 0;
            break;
        case 'n':
            if (int (sqrt(atoi(optarg))) * int (sqrt(atoi(optarg))) != atoi(optarg) )
                return(-1);
            nRows = sqrt(atoi(optarg));
            nColumns = sqrt(atoi(optarg));
            break;
        case 'x':
            nRows = atoi(optarg);
            break;
        case 'y':
            nColumns = atoi(optarg);
      }

    // total number of neurons in the SOM
    int nNeurons = nRows * nColumns;
    // total length of the serialized matrix
    int totalLength = nRows * nColumns * nElements;

    // host SOM
    float *h_Matrix = (float *)malloc(sizeof(float) * totalLength);
    // host sample array
    float *h_Sample = (float *)malloc(sizeof(float) * nElements);
    // host distance array, used to find BMU
    float *h_Distance = (float *) malloc(sizeof(float) * nNeurons);

    //random SOM initialization
    for(int i = 0; i < totalLength; i++){
    	h_Matrix[i] = i;
    }
    // distance array inizialization
    for(int i = 0; i < nNeurons; i++){
    	h_Distance[i] = 0;
    }

    //random sample inizialization, used for TEST
    for(int i = 0; i < nElements; i++){
    	h_Sample[i] = i+1;
    }

    // device SOM
    float *d_Matrix;
    // device sample array
    float *d_Sample;
    // device distance array, 
    float *d_Distance;

    //device malloc
    hipMalloc((void **)&d_Matrix, sizeof(float) * totalLength);
    hipMalloc((void**)&d_Sample, sizeof(float) * nElements);
    hipMalloc((void**)&d_Distance, sizeof(float) * nNeurons);

	//copy from host to device matrix, sample and distance
	hipMemcpy(d_Matrix, h_Matrix, sizeof(float) * totalLength, hipMemcpyHostToDevice);
	hipMemcpy(d_Sample, h_Sample, sizeof(float) * nElements, hipMemcpyHostToDevice);
	hipMemcpy(d_Distance, h_Distance, sizeof(float) * nNeurons, hipMemcpyHostToDevice);	
	
    //peparing param to launch kernel
    int nblocks = (nNeurons / 1024) + 1; 
    compute_distance<<<nblocks,1024>>>(d_Matrix, nNeurons, d_Sample, d_Distance, nElements);

	//wait for all block to be completed
    hipDeviceSynchronize();

    /*
    hipMemcpy(h_Distance, d_Distance, sizeof(float) * nNeurons, hipMemcpyDeviceToHost);
    
    for(int i = 0; i < nNeurons; i++){
        std::cout << h_Distance[i] << std::endl;
    }
    */

	//create thrust vector to find BMU
	thrust::device_vector<float> d_vec_Distance(d_Distance, d_Distance + nNeurons);
	//extract the first element
	thrust::device_vector<float>::iterator iter = thrust::min_element(d_vec_Distance.begin(), d_vec_Distance.end());
	// find index of BMU
	unsigned int BMU_index = iter - d_vec_Distance.begin();
	float BMU_value = *iter;

	std::cout << "The minimum value is " << BMU_value << " at position " << BMU_index << std::endl;
	//TODO: update BMU and neighbors
 
    hipFree(d_Matrix);
    free(h_Matrix);

}

