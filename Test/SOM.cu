#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void
test_kernel(float* matrix, int numElements)
{

	matrix[blockIdx.x] = matrix[blockIdx.x] * 2;
}

int
main(void)
{

    int numberElements = 14;
    int numberRows = 10;
    int numberColoumns = 10;
    int totalLength = numberRows * numberColoumns * numberElements;

    float *hostMatrix = (float *)malloc(sizeof(float) * totalLength);
    for(int i = 0; i < totalLength; i++){
    	hostMatrix[i] = i;
    	std::cout << hostMatrix[i] << std::endl;
    }

    float *deviceMatrix;
    hipMalloc((void **)&deviceMatrix, sizeof(float) * totalLength);

	//copy to device memory
	hipMemcpy(deviceMatrix, hostMatrix, sizeof(float) * totalLength, hipMemcpyHostToDevice);

	test_kernel<<<totalLength,1>>>(deviceMatrix, totalLength);

    hipDeviceSynchronize();

	hipMemcpy(hostMatrix, deviceMatrix, sizeof(float) * totalLength, hipMemcpyDeviceToHost);

    for(int i = 0; i < totalLength; i++){
    	std::cout << hostMatrix[i] << std::endl;
    }

    hipFree(deviceMatrix);
    free(hostMatrix);

}

