#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <unistd.h>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <iostream>
#include <random>
#include <float.h>

#include "utility_functions.cpp"
#include "distance_kernels.cu"
#include "cmdline.h"

#define CUDA_CHECK_RETURN(value) {											\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(-13);															\
} }

int main(int argc, char **argv)
{
	// reading passed params
	gengetopt_args_info ai;
    if (cmdline_parser (argc, argv, &ai) != 0)
    {
        exit(1);
    }
	// INIZIALIZING VARIABLES WITH DEFAULT VALUES
	// path of the input file
	std::string filePath = ai.inputfile_arg;
	// verbose flag
    bool verbose = ai.verbose_flag;
    // advanced debug flag
    bool debug = ai.debug_flag;
    // save SOM to file
    bool print = ai.save_flag;
    // number of rows in the martix
    int nRows = ai.nRows_arg;
    // number of column in the martix
    int nColumns = ai.nColumns_arg;
    // initial learning rate
    double ilr = ai.initial_learning_rate_arg;
    // final learning rate
    double flr = ai.final_learning_rate_arg;
    // max number of iteration
    int maxnIter = ai.iteration_arg;
    // accuracy threshold
    double accuracyTreshold = ai.accuracy_arg;
    // Initial radius of the update
    double initialRadius = ai.radius_arg;
    // type of distance used
    char distanceType = ai.distance_arg[0];
    // enable the normalization of the distance fuction
    bool normalizeFlag = ai.normalize_flag;
    // type of neighbors function used
    char neighborsType = ai.neighbors_arg[0];
    // type of initialization
    char initializationType = ai.initialization_arg[0];
    // type of lactice used
    char lacticeType = ai.lactice_arg[0];
    // exponential decay for radius and lr
    char exponential = ai.exponential_arg[0];
    // dataset presentation methon
    bool randomizeDataset = ai.randomize_flag;
    // counter for times of Samples vector is presented to the SOM
    int nIter = 0;
    // declaration of some usefull variables
    double min_neuronValue, max_neuronValue;
    // number of lines in the input file
    int nSamples;
    // total number of neurons in the SOM
    int nNeurons;
    // total length of the matrix vector
    int totalLength;
    // number of features per read
    int nElements;
    // number of blocks that needs to be launched
    int nblocks;
    // actual learning rate
    double lr;
    // actual radius
    double radius;
    // actual accuracy
	double accuracy;

    // READ THE INPUT FILE
    // vector of samples to be analized from the SOM
    std::vector <double> Samples;
    // retrive the number of features readed from the file
    nElements = readSamplesfromFile(Samples, filePath);

    // EXTRACTING THE MIN/MAX FROM SAMPLES(only used for random initialization)
    if (initializationType == 'r')
    {
	    // creating the thrust vector
	    thrust::device_vector<double> t_Samples(Samples);
	    // extract the minimum
	    thrust::device_vector<double>::iterator it = thrust::min_element(t_Samples.begin(), t_Samples.end());
	    min_neuronValue = *it;
	    // extract maximum
	    thrust::device_vector<double>::iterator it2 = thrust::max_element(t_Samples.begin(), t_Samples.end());
	    max_neuronValue = *it2;
	}

    // COMPUTE VALUES FOR THE SOM INITIALIZATION
    // retrive the number of samples
    nSamples = Samples.size() / nElements;

    // estimate the neurons number if not given
    if (nRows ==0 | nColumns == 0)
    {
    	int tmp = 5*(pow(nSamples, 0.54321));
    	nRows = sqrt(tmp);
    	nColumns = sqrt(tmp);
    }

    // estimate the radius if not given (covering 2/3 of the matrix)
    if (initialRadius == 0)
    	initialRadius = 1 + (max(nRows, nColumns)/2) * 2 / 3;

    // total number of neurons in the SOM
    nNeurons = nRows * nColumns;
    // total length of the serialized matrix
    totalLength = nRows * nColumns * nElements;
    
    // number of block used in the computation
    nblocks = (nNeurons / getnThreads()) + 1;

    // CHECKING COMPUTABILITY ON CUDA
    if (nblocks >= 65535)
    {
    	std::cout << "Too many bocks generated, cannot run a kernel with so many blocks. Try to reduce the number of neurons" << std::endl;
    	exit(-1);
    }

    // CHECK AVAILABLE MEMORY
    if (sizeof(double) * nNeurons * nElements >= checkFreeGpuMem())
    {
	    	std::cout << "Not enougth memory on the GPU, try to reduce neurons' number" << std::endl;
	    	exit(-1);
	}

    // ALLOCATION OF THE STRUCTURES
    // host SOM
    double *h_Matrix = (double *)malloc(sizeof(double) * totalLength);
    // host sample array
    double *h_ActualSample = (double *)malloc(sizeof(double) * nElements);
    // host distance array, used to find BMU
    double *h_Distance = (double *) malloc(sizeof(double) * nNeurons);
    // device SOM
    double *d_Matrix;
    // device sample array
    double *d_ActualSample;
    // device distance array, 
    double *d_Distance;
    // device malloc
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_Matrix, sizeof(double) * totalLength));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_ActualSample, sizeof(double) * nElements));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_Distance, sizeof(double) * nNeurons));

    // SOM INIZIALIZATION
    // generating random seed
    std::random_device rd;
    std::mt19937 e2(rd());
    if (initializationType == 'r')
    {
    	std::uniform_real_distribution<> dist(min_neuronValue, max_neuronValue);
	    for(int i = 0; i < totalLength; i++)
	    {
	    	h_Matrix[i] = dist(e2); 
	    }
    }
    else if (initializationType == 'c')
    {
    	std::uniform_int_distribution<> dist(0, nSamples);
	    for (int i = 0; i < nNeurons; i++)
	    {
	        int r = dist(e2);
	        for (int k = i * nElements, j = 0; j < nElements; k++, j++)
	        {
	             h_Matrix[k] = Samples[r*nElements + j];
	        }
	    }
	}
	else
    {
		// TODO PCA
	}

    if (debug | print)
        saveSOMtoFile("initialSOM.out", h_Matrix, nRows, nColumns, nElements);
    
	// inizializing actual values
    lr = ilr;
    radius = initialRadius;
	accuracy = DBL_MAX;

    // debug print
    if(verbose | debug)
    {
        std::cout << "Running the program with " << nRows  << " rows, " << nColumns << " columns, " << nNeurons << " neurons, " << nElements << " features fot each read, " << ilr << " initial learning rate, " << flr << " final learning rate, " << accuracyTreshold<< " required accuracyTreshold, " << radius << " initial radius, ";
        std::cout << maxnIter << " max total iteration, " << distanceType << " distance type, " << normalizeFlag << " normalized, " << neighborsType << " neighbors function, ";
        std::cout << initializationType << " initialization teqnique, " << lacticeType << " lactice, " << exponential << " type of decay, " << randomizeDataset << " randomized input, " << nSamples << " sample in the input file" << nblocks << "blocks will be launched on the GPU" << std::endl;
    
    }

    // initializing indexes to shuffle the Samples vector
    int randIndexes[nSamples];
    for (int i = 0; i < nSamples; i++)
    {
    	randIndexes[i] = i;
    }

    // thrust vector used to store the BMU distances of each iteration
    thrust::device_vector<double> d_DistanceHistory;

    while((accuracy >= accuracyTreshold) && (lr >= flr) && (nIter < maxnIter))
    {
    	// randomize indexes of samples
    	if(randomizeDataset)
    		std::random_shuffle(&randIndexes[0], &randIndexes[nSamples-1]);

        if (debug | verbose)
        {
            std::cout << "Learning rate of this iteration is " << lr << std::endl;
            std::cout << "Radius of this iteration is " << radius << std::endl;
        }

        // ITERATE ON EACH SAMPLE TO FIND BMU
	    for(int s=0; s < nSamples ; s++)
        {

		    // copy the s sample in the actual sample vector
		    for(int i = randIndexes[s]*nElements, j = 0; i < randIndexes[s]*nElements+nElements; i++, j++)
            {
		    	h_ActualSample[j] = Samples[i];
		    } 

			// copy from host to device matrix, actual sample and distance
			CUDA_CHECK_RETURN(hipMemcpy(d_Matrix, h_Matrix, sizeof(double) * totalLength, hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy(d_ActualSample, h_ActualSample, sizeof(double) * nElements, hipMemcpyHostToDevice));
			
		    // parallel search launch
		    if (normalizeFlag)
            {
		    	switch(distanceType)
                {
		    		case 'e' : compute_distance_euclidean_normalized<<<nblocks, 1024>>>(d_Matrix, d_ActualSample, d_Distance, nNeurons, nElements); break;
		    		case 's' : compute_distance_sum_squares_normalized<<<nblocks, 1024>>>(d_Matrix, d_ActualSample, d_Distance, nNeurons, nElements); break;
		    		case 'm' : compute_distance_manhattan_normalized<<<nblocks, 1024>>>(d_Matrix, d_ActualSample, d_Distance, nNeurons, nElements); break;
		    		case 't' : compute_distance_tanimoto<<<nblocks, 1024>>>(d_Matrix, d_ActualSample, d_Distance, nNeurons, nElements); break;
		    	}
		    }
            else
            {
		    	switch(distanceType)
                {
		    		case 'e' : compute_distance_euclidean<<<nblocks, 1024>>>(d_Matrix, d_ActualSample, d_Distance, nNeurons, nElements); break;
		    		case 's' : compute_distance_sum_squares<<<nblocks, 1024>>>(d_Matrix, d_ActualSample, d_Distance, nNeurons, nElements); break;
		    		case 'm' : compute_distance_manhattan<<<nblocks, 1024>>>(d_Matrix, d_ActualSample, d_Distance, nNeurons, nElements); break;
				    case 't' : compute_distance_tanimoto<<<nblocks, 1024>>>(d_Matrix, d_ActualSample, d_Distance, nNeurons, nElements); break;
		    	}
		    }

			//wait for all block to complete the computation
		    hipDeviceSynchronize();

		    // CHECK AVAILABLE MEMORY
	    	if (sizeof(double) * nNeurons * nElements >= checkFreeGpuMem())
            {
		    	std::cout << "Out of memory, try to reduce the neurons number" << std::endl;
		    	exit(-1);
			}
				
			// create thrust vector to find BMU  in parallel
			thrust::device_vector<double> d_vec_Distance(d_Distance, d_Distance + nNeurons);
			// extract the first matching BMU
			thrust::device_vector<double>::iterator iter = thrust::min_element(d_vec_Distance.begin(), d_vec_Distance.end());
			// extract index and value of BMU
			unsigned int BMU_index = iter - d_vec_Distance.begin();
            unsigned int BMU_x = BMU_index / nColumns;
            unsigned int BMU_y = BMU_index % nColumns;
			double BMU_distance = *iter;

            double tmp = 0;
            double dist = 0;
            for(int u = 0; u < nElements; u++)
            {
                tmp = h_Matrix[BMU_index*nElements + u] - h_ActualSample[u];
                dist += tmp * tmp; 
            }


			// adding the found value in the distance history array
			d_DistanceHistory.push_back(dist);

			// debug print
		    if(debug)
			   std::cout << "The minimum distance is " << dist << " at position " << BMU_index << std::endl;

			// UPDATE THE NEIGHBORS
			// if radius is 0, update only BMU 
	        if (radius == 0)
	        {
	        	for (int i = BMU_index * nElements, j = 0; j < nElements; i++, j++)
                {
	        		h_Matrix[i] = h_Matrix[i] + lr * (h_ActualSample[j] - h_Matrix[i]);
	        	}
	        }
	        // possible to transfer on the gpu?
            // update also the neighbors
	        else
	        {
	            for (int i = 0; i < nNeurons; i++){
	                int x = i / nColumns;
	                int y = i % nColumns;
                    int distance = 0;
                    if (lacticeType == 's')
	                   distance = sqrt((x - BMU_x) * (x - BMU_x) + (y - BMU_y) * (y - BMU_y));
                    else
                        distance = ComputeDistanceHexGrid(BMU_x, BMU_y, x, y);
                    
                    // update only if...
	                if (distance <= radius)
                    {
                        double neigh = 0.0;
                        switch (neighborsType)
                        {
                            case 'g' : neigh = gaussian(distance, radius); break;
                            case 'b' : neigh = bubble(distance, radius); break;
                            case 'm' : neigh = mexican_hat(distance, radius); break;
                        } 

	                    for (int k = i * nElements, j = 0; j < nElements; k++, j++)
                        {
	                        h_Matrix[k] = h_Matrix[k] + neigh * lr * (h_ActualSample[j] - h_Matrix[k]);
	                    }
	                }
	            }
        	}	         
        }

        // END OF SAMPLES ITERATION. UPDATING VALUES
        // updating accuracy
        //accuracy = thrust::reduce(d_DistanceHistory.begin(),d_DistanceHistory.end())/ ((double)nSamples);
        //d_DistanceHistory.clear();
        accuracy = thrust::reduce(d_DistanceHistory.begin(), d_DistanceHistory.end());
        accuracy = sqrt(accuracy/nElements)/nSamples;
        d_DistanceHistory.clear();

        if (verbose | debug)
        {
            std::cout << "Mean distance of this iteration is " << accuracy << std::endl;
        }
        
		// updating the counter iteration
		nIter ++;

        // updating radius and learning rate
        radius = (int) (initialRadius - (initialRadius) * ((double)nIter/maxnIter));
        if (exponential== 'r' | exponential == 'b')
        	radius = (int) (initialRadius * exp(-(double)nIter/(sqrt(maxnIter)))) ;

        lr = ilr - (ilr - flr) * ((double)nIter/maxnIter);
        if (exponential== 'l' | exponential == 'b')
        	lr = ilr * exp(- (double)nIter/sqrt(maxnIter)) + flr;
    }

    if (debug | print)
    {
        saveSOMtoFile("outputSOM.out", h_Matrix, nRows, nColumns, nElements);
    }

	//freeing all allocated memory
    hipFree(d_Matrix);
    hipFree(d_ActualSample);
    hipFree(d_Distance);
    free(h_Matrix);
    free(h_Distance);
    free(h_ActualSample);
}