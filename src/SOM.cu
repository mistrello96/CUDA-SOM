#include "hip/hip_runtime.h"
#include <ctime>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <unistd.h>
#include <fstream>
#include <sstream>
#include <float.h>
#include <algorithm>
#include <iostream>

#include "utility_functions.cpp"
#include "distance_kernels.cu"
#include "cmdline.h"

#define CUDA_CHECK_RETURN(value) {											\
		hipError_t _m_cudaStat = value;										\
		if (_m_cudaStat != hipSuccess) {										\
			fprintf(stderr, "Error %s at line %d in file %s\n",					\
					hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
					exit(-13);															\
} }

int main(int argc, char **argv)
{
	// reading passed params
	gengetopt_args_info ai;
    if (cmdline_parser (argc, argv, &ai) != 0) {
        exit(1);
    }
	// INIZIALIZING VARIABLES WITH DEFAULT VALUES
	// path of the input file
	std::string filePath = ai.inputfile_arg;
	// verbose flag
    bool verbose = ai.verbose_flag;
    // advanced debug flag
    bool debug = ai.debug_flag;
    // save SOM to file
    bool print = ai.save_flag;
    // number of rows in the martix
    int nRows = ai.nRows_arg;
    // number of column in the martix
    int nColumns = ai.nColumns_arg;
    // initial learning rate
    double ilr = ai.initial_learning_rate_arg;
    // final learning rate
    double flr = ai.final_learning_rate_arg;
    // max number of iteration
    int maxnIter = ai.iteration_arg;
    // accuracy threshold
    double accuracyTreshold = ai.accuracy_arg;
    // counter for times of Samples vector is presented to the SOM
    int nIter = 0;
    // Initial radius of the update
    double initialRadius = ai.radius_arg;
    // type of distance used
    char distanceType = ai.distance_arg[0];
    // enable the normalization of the distance fuction
    bool normalizeFlag = ai.normalize_flag;
    // type of neighbors function used
    char neighborsType = ai.neighbors_arg[0];
    // type of initialization
    char initializationType = ai.initialization_arg[0];
    // type of lactice used
    char lacticeType = ai.lactice_arg[0];
    // dataset presentation methon
    bool randomizeDataset = ai.randomize_flag;
    // declaration of some usefull variables
    double min_neuronValue, max_neuronValue;
    int nSamples;
    int nNeurons;
    int totalLength;
    int nblocks;
    double lr;
    double radius;
	double accuracy;

    // READ THE INPUT FILE
    // vector of samples to be analized from the SOM
    std::vector <double> Samples;
    // retrive the number of features readed from the file
    int nElements = readSamplesfromFile(Samples, filePath);

    // EXTRACTING THE MIN/MAX FROM SAMPLES
    if (initializationType == 'r'){
	    // creating the thrust vector
	    thrust::device_vector<double> t_Samples(Samples);
	    // extract the minimum
	    thrust::device_vector<double>::iterator it = thrust::min_element(t_Samples.begin(), t_Samples.end());
	    min_neuronValue = *it;
	    // extract maximum
	    thrust::device_vector<double>::iterator it2 = thrust::max_element(t_Samples.begin(), t_Samples.end());
	    max_neuronValue = *it2;
	}

    // COMPUTE VALUES FOR THE SOM INITIALIZATION
    // retrive the number of samples
    nSamples = Samples.size() / nElements;

    // estimate the neurons number if not given
    if (nRows ==0 | nColumns == 0)
    {
    	int tmp = 5*(pow(nSamples, 0.54321));
    	nRows = sqrt(tmp);
    	nColumns = sqrt(tmp);
    }

    // estimate the radius if not given (covering 2/3 of the matrix)
    if (initialRadius == 0)
    	initialRadius = 1 + (max(nRows, nColumns)/2) * 2 / 3;

    // total number of neurons in the SOM
    nNeurons = nRows * nColumns;
    // total length of the serialized matrix
    totalLength = nRows * nColumns * nElements;
    // number of block used in the computation

    nblocks = (nNeurons / getnThreads()) + 1;

    // CHECKING COMPUTABILITY ON CUDA
    if (nblocks >= 65535)
    {
    	std::cout << "Too many bocks generated, cannot run a kernel with so many blocks. Try to reduce the number of neurons" << std::endl;
    	exit(-1);
    }

    // CHECK AVAILABLE MEMORY
    if (sizeof(double) * nNeurons * nElements >= checkFreeGpuMem())
    {
	    	std::cout << "Not enougth memory on the GPU, try to reduce neurons' number" << std::endl;
	    	exit(-1);
	}

    // ALLOCATION OF THE STRUCTURES
    // host SOM
    double *h_Matrix = (double *)malloc(sizeof(double) * totalLength);
    // host sample array
    double *h_ActualSample = (double *)malloc(sizeof(double) * nElements);
    // host distance array, used to find BMU
    double *h_Distance = (double *) malloc(sizeof(double) * nNeurons);
    // host BMU distance array
    double *h_DistanceHistory = (double *)malloc(sizeof(double) * nSamples);
    // device SOM
    double *d_Matrix;
    // device sample array
    double *d_ActualSample;
    // device distance array, 
    double *d_Distance;
    // device malloc
    CUDA_CHECK_RETURN(hipMalloc((void **)&d_Matrix, sizeof(double) * totalLength));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_ActualSample, sizeof(double) * nElements));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_Distance, sizeof(double) * nNeurons));

    // SOM INIZIALIZATION
    // generating random seed
    srand(time(NULL));
    if (initializationType == 'r'){
	    for(int i = 0; i < totalLength; i++)
	    {
	        double tmp = rand() / (float) RAND_MAX;
	    	tmp = min_neuronValue + tmp * (max_neuronValue - min_neuronValue);
	    	h_Matrix[i] = tmp; 
	    }
    }
    else if (initializationType == 'c'){
	    for (int i = 0; i < nNeurons; i++)
	    {
	        int r = rand() % nSamples;
	        for (int k = i * nElements, j = 0; j < nElements; k++, j++)
	        {
	             h_Matrix[k] = Samples[r*nElements + j];
	        }
	    }
	}
	else {
		// TODO PCA
	}

    if (debug | print){
        saveSOMtoFile("initialSOM.out", h_Matrix, nRows, nColumns, nElements);
    }
	// inizializing the learnig rate
    lr = ilr;
    // initializiang the radius of the updating function
    radius = initialRadius;
    // initializing accuracy of the first iteration with a fake value
	accuracy = DBL_MAX;

    // debug print
    if(verbose | debug){
        std::cout << "Running the program with " << nRows  << " rows, " << nColumns << " columns, " << nNeurons << " neurons, " << nElements << " features fot each read, " << ilr << " initial learning rate, " << flr << " final learning rate, " << accuracyTreshold<< " required accuracyTreshold, " << radius << " initial radius, "  << std::endl;
    }

    // initializing indexes to shuffle the Samples vector
    int randIndexes[nSamples];
    for (int i = 0; i < nSamples; i++)
    {
    	randIndexes[i] = i;
    }

    while((accuracy >= accuracyTreshold) && (lr >= flr) && (nIter < maxnIter)){
    	// randomize indexes of samples
    	if(randomizeDataset)
    		std::random_shuffle(&randIndexes[0], &randIndexes[nSamples-1]);

        if (debug | verbose)
        {
            std::cout << "Learning rate of this iteration is " << lr << std::endl;
            std::cout << "Radius of this iteration is " << radius << std::endl;
        }

        // ITERATE ON EACH SAMPLE TO FIND BMU
	    for(int s=0; s < nSamples ; s++){

		    // distance array inizialization
		    for(int i = 0; i < nNeurons; i++){
		    	h_Distance[i] = 0;
		    }

		    // copy the s sample in the actual sample vector
		    for(int i = randIndexes[s]*nElements, j = 0; i < randIndexes[s]*nElements+nElements; i++, j++){
		    	h_ActualSample[j] = Samples[i];
		    } 

			// copy from host to device matrix, actual sample and distance
			CUDA_CHECK_RETURN(hipMemcpy(d_Matrix, h_Matrix, sizeof(double) * totalLength, hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy(d_ActualSample, h_ActualSample, sizeof(double) * nElements, hipMemcpyHostToDevice));
			CUDA_CHECK_RETURN(hipMemcpy(d_Distance, h_Distance, sizeof(double) * nNeurons, hipMemcpyHostToDevice));	
			
		    // parallel search launch
		    compute_distance_euclidean_normalized<<<nblocks,1024>>>(d_Matrix, d_ActualSample, d_Distance, nNeurons, nElements);

			//wait for all block to complete the computation
		    hipDeviceSynchronize();

		    // CHECK AVAILABLE MEMORY
	    	if (sizeof(double) * nNeurons * nElements >= checkFreeGpuMem()){
		    	std::cout << "Out of memory, try to reduce the neurons number" << std::endl;
		    	exit(-1);
			}
				
			// create thrust vector to find BMU  in parallel
			thrust::device_vector<double> d_vec_Distance(d_Distance, d_Distance + nNeurons);
			// extract the first matching BMU
			thrust::device_vector<double>::iterator iter = thrust::min_element(d_vec_Distance.begin(), d_vec_Distance.end());
			// extract index and value of BMU
			unsigned int BMU_index = iter - d_vec_Distance.begin();
            unsigned int BMU_x = BMU_index / nColumns;
            unsigned int BMU_y = BMU_index % nColumns;
			double BMU_distance = *iter;
			// adding the found value in the distance history array
			h_DistanceHistory[s] = BMU_distance;

			// debug print
		    if(debug)
			   std::cout << "The minimum distance is " << BMU_distance << " at position " << BMU_index << std::endl;

			// UPDATE THE NEIGHBORS
			// if radius is 0, update only BMU 
	        if (radius == 0)
	        {
	        	for (int i = BMU_index * nElements, j = 0; j < nElements; i++, j++){
	        		h_Matrix[i] = h_Matrix[i] + lr * (h_ActualSample[j] - h_Matrix[i]);
	        	}
	        }
	        // possible to transfer on the gpu
	        else
	        {
	            for (int i = 0; i < nNeurons; i++){
	                int x = i / nColumns;
	                int y = i % nColumns;
	                int distance = sqrt((x - BMU_x)*(x - BMU_x) + (y - BMU_y)*(y - BMU_y));
	                if (distance <= radius){
	                    double g = gaussian(distance, radius);
	                    int b = bubble(distance, radius);
	                    for (int k = i * nElements, j = 0; j < nElements; k++, j++){
	                        h_Matrix[k] = h_Matrix[k] + g * lr * (h_ActualSample[j] - h_Matrix[k]);
	                    }
	                }
	            }
        	}	         
		
        }

        // END OF SAMPLES ITERATION. UPDATING VALUES
        // updating accuracy
        thrust::device_vector<double> d_DistanceHistory(h_DistanceHistory, h_DistanceHistory + nSamples);
        double meansum = thrust::reduce(d_DistanceHistory.begin(), d_DistanceHistory.end());
        accuracy = meansum / ((double)nSamples);
        if (verbose | debug)
        {
            std::cout << "Mean distance of this iteration is " << accuracy << std::endl;
        }

		// updating the counter iteration
		nIter ++;
        // updating radius and learning rate
        radius =(int) (initialRadius - (initialRadius) * ((double)nIter/maxnIter));
        lr = ilr - (ilr - flr) * ((double)nIter/maxnIter);
    }

    if (debug | print){
        saveSOMtoFile("outputSOM.out",h_Matrix, nRows, nColumns, nElements);
    }

	//freeing all allocated memory
    hipFree(d_Matrix);
    hipFree(d_ActualSample);
    hipFree(d_Distance);
    free(h_Matrix);
    free(h_Distance);
    free(h_ActualSample);

}

