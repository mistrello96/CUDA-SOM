#include "hip/hip_runtime.h"
// kernel to update SOM after the BMU has been found. Called only if radius of the update is 0, so only BMU will be updated
__global__ void update_BMU(double* k_Matrix, double* k_Samples, double lr, int samplesIndex, int nElements, int BMUIndex)
{
	// update all features of the BMU
    for (int i = BMUIndex * nElements, j=0; j < nElements; i++, j++)
    {
        k_Matrix[i] = k_Matrix[i] + lr * (k_Samples[samplesIndex + j] - k_Matrix[i]); 
    }
}

// kernel to update SOM after the BMU has been found. Called when radius is > 0, all the SOM neurons will be updated. 
__global__ void update_SOM(double* k_Matrix, double* k_Samples, double lr, int samplesIndex, int nElements, int BMUIndex, int nColumns, int radius, int nNeuron, char neighborsType)
{
	// compute neuron's index
    int threadindex = threadIdx.x + blockDim.x * blockIdx.x;
    if (threadindex < nNeuron){
        // compute distance if lattice is square
        int distance = sqrtf(((threadindex / nColumns) - (BMUIndex / nColumns)) * ((threadindex / nColumns) - (BMUIndex / nColumns)) + ((threadindex % nColumns) - (BMUIndex % nColumns)) * ((threadindex % nColumns) - (BMUIndex % nColumns)));
        if (distance <= radius)
        {
            double neigh = 0;
            // compute neigh param as requested
            switch (neighborsType)
            {
                case 'g' : neigh = gaussian(distance, radius); break;
                case 'b' : neigh = bubble(distance, radius); break;
                case 'm' : neigh = mexican_hat(distance, radius); break;
            }
            // update all features of the neuron
            for (int i = threadindex * nElements, j=0; j < nElements; i++,j++)
            {
                k_Matrix[i] = k_Matrix[i] + neigh * lr * (k_Samples[samplesIndex + j] - k_Matrix[i]);
            }
        }
    }
}

// kernel to update SOM after the BMU has been found. Called when radius is > 0, all the SOM neurons will be updated. 
__global__ void update_SOM_toroidal(double* k_Matrix, double* k_Samples, double lr, int samplesIndex, int nElements, int BMUIndex, int nRows, int nColumns, int radius, int nNeuron, char neighborsType)
{
	// compute neuron's index
    int threadindex = threadIdx.x + blockDim.x * blockIdx.x;
    if (threadindex < nNeuron){
        // call function to compute distance in a toroidal square map
        int distance = ComputeDistanceToroidal(threadindex / nColumns, threadindex % nColumns, BMUIndex / nColumns, BMUIndex % nColumns, nRows, nColumns);
        if (distance <= radius)
        {
            double neigh = 0;
            // compute neigh param as requested
            switch (neighborsType)
            {
                case 'g' : neigh = gaussian(distance, radius); break;
                case 'b' : neigh = bubble(distance, radius); break;
                case 'm' : neigh = mexican_hat(distance, radius); break;
            }
            // update all features of the neuron
            for (int i = threadindex * nElements, j=0; j < nElements; i++,j++)
            {
                k_Matrix[i] = k_Matrix[i] + neigh * lr * (k_Samples[samplesIndex + j] - k_Matrix[i]);
            }
        }
    }
}

// kernel to update a exagonal SOM after the BMU has been found. Called when radius is > 0, all the SOM neurons will be updated. 
__global__ void update_SOM_exagonal(double* k_Matrix, double* k_Samples, double lr, int samplesIndex, int nElements, int BMUIndex, int nColumns, int radius, int nNeuron, char neighborsType)
{
	// compute neuron's index
    int threadindex = threadIdx.x + blockDim.x * blockIdx.x;
    if (threadindex < nNeuron){
        // call function to compute distance in a exagonal map
        int distance = ComputeDistanceHexGrid(BMUIndex / nColumns, BMUIndex % nColumns, threadindex / nColumns, threadindex % nColumns);
        if (distance <= radius)
        {
            double neigh =0;
            // compute neigh param as requested
            switch (neighborsType)
            {
                case 'g' : neigh = gaussian(distance, radius); break;
                case 'b' : neigh = bubble(distance, radius); break;
                case 'm' : neigh = mexican_hat(distance, radius); break;
            }
            // update all features of the neuron
            for (int i = threadindex * nElements, j=0; j < nElements; i++,j++)
            {
                k_Matrix[i] = k_Matrix[i] + neigh * lr * (k_Samples[samplesIndex + j] - k_Matrix[i]);
            }
        }
    }
}

// kernel to update a exagonal toroidal SOM after the BMU has been found. Called when radius is > 0, all the SOM neurons will be updated. 
__global__ void update_SOM_exagonal_toroidal(double* k_Matrix, double* k_Samples, double lr, int samplesIndex, int nElements, int BMUIndex, int nRows, int nColumns, int radius, int nNeuron, char neighborsType)
{
    // compute neuron's index
    int threadindex = threadIdx.x + blockDim.x * blockIdx.x;
    if (threadindex < nNeuron){
        // call function to compute distance in a toroidal exagonal map
        int distance = ComputeDistanceHexGridToroidal(BMUIndex / nColumns, BMUIndex % nColumns, threadindex / nColumns, threadindex % nColumns, nRows, nColumns);
        if (distance <= radius)
        {
            double neigh =0;
            // compute neigh param as requested
            switch (neighborsType)
            {
                case 'g' : neigh = gaussian(distance, radius); break;
                case 'b' : neigh = bubble(distance, radius); break;
                case 'm' : neigh = mexican_hat(distance, radius); break;
            }
            // update all features of the neuron
            for (int i = threadindex * nElements, j=0; j < nElements; i++,j++)
            {
                k_Matrix[i] = k_Matrix[i] + neigh * lr * (k_Samples[samplesIndex + j] - k_Matrix[i]);
            }
        }
    }
}